#include "hip/hip_runtime.h"
#include <torch/types.h>
#include <iostream>
#include <math.h>
#include <limits>
#include "hip/hip_runtime.h"
#include "gpu.cuh"

__global__ void quantize(int num_quant_levels,
                         float* quant_levels,
                         int num_elements,
                         float* tensor,
                         float* result) {

	int index = blockIdx.x * blockDim.x + threadIdx.x;
	int stride = blockDim.x * gridDim.x;

	for (int i = index; i < num_elements; i += stride) {
		int middle_point; // Middle point
		int optimal_point; // Optimal point
		int l = 0; // Lower bound
		int h = num_quant_levels; // Higher bound
		float difference = 1.0f; // Difference between a given point and the current middle point
		while (l <= h) {
			middle_point = l + (h - l) / 2;
			if (abs(tensor[i] - quant_levels[middle_point]) < difference) {
				// If the distance between the new point is smaller than the current distance
				difference = abs(tensor[i] - quant_levels[middle_point]);
				optimal_point = middle_point;
			}
			if (quant_levels[middle_point] < tensor[i]) {
				l = middle_point + 1;
			}
			else {
				h = middle_point - 1;
			}
		}
		result[i] = quant_levels[optimal_point];
	}
	return;
}

torch::Tensor quant_cuda(torch::Tensor tensor, int num_quant_levels, float min_value, float max_value) {
  // Determine quantization levels
  torch::Tensor quant_levels = at::linspace(min_value, max_value, num_quant_levels);
  torch::Tensor result = torch::zeros(tensor.sizes());
  float *quant_levels_gpu;
  torch::Tensor *result_gpu;
  hipMalloc(&quant_levels_gpu, sizeof(float) * quant_levels.numel());
  hipMalloc(&result_gpu, sizeof(float) * tensor.numel());
  hipMemcpy(quant_levels_gpu, quant_levels.data<float>(), sizeof(float) * quant_levels.numel(), hipMemcpyHostToDevice);
  // hipMemcpy(result_gpu, result.data<float>(), sizeof(float) * tensor.numel(), hipMemcpyHostToDevice);
  quantize<<<GET_BLOCKS(tensor.numel()), CUDA_NUM_THREADS>>>(num_quant_levels, quant_levels_gpu, tensor.numel(), tensor.data<float>(), result_gpu.data<float>());
  hipDeviceSynchronize();
  hipFree(quant_levels_gpu);
  return result;
}
